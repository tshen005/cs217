/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.cu"

#define  StreamN   3




int main (int argc, char *argv[])
{
    //set standard seed
    srand(217);

    Timer timer;
    
    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    size_t A_sz, B_sz, C_sz;
    unsigned int VecSize;
   

    if (argc == 1) {
        VecSize = 1000000;

    } else if (argc == 2) {
        VecSize = atoi(argv[1]);      
    }
  
    else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
    }

    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);

    if(!prop.deviceOverlap){
        printf("No device will handle overlaps. so no speed up from stream.\n");
        return 0;
    }

    A_sz = VecSize;
    B_sz = VecSize;
    C_sz = VecSize;

    const unsigned int BLOCK_SIZE = 512;
    const unsigned int SegSize = VecSize / StreamN;
    unsigned int leftover = VecSize % (SegSize * StreamN);

    hipStream_t stream0, stream1, stream2;
    hipStreamCreateWithFlags(&stream0, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);


    hipHostAlloc((void**)&A_h, A_sz*sizeof(float),hipHostMallocDefault);
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    hipHostAlloc((void**)&B_h, B_sz*sizeof(float),hipHostMallocDefault);
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    hipHostAlloc((void**)&C_h, C_sz*sizeof(float),hipHostMallocDefault);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    size Of vector: %u x %u\n  ", VecSize);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    float *A_d0, *B_d0, *C_d0;
    float *A_d1, *B_d1, *C_d1;
    float *A_d2, *B_d2, *C_d2;
   
    //INSERT CODE HERE

    hipMalloc((void**)&A_d0, sizeof(float)*SegSize);
    hipMalloc((void**)&B_d0, sizeof(float)*SegSize);
    hipMalloc((void**)&C_d0, sizeof(float)*SegSize);
    hipMalloc((void**)&A_d1, sizeof(float)*SegSize);
    hipMalloc((void**)&B_d1, sizeof(float)*SegSize);
    hipMalloc((void**)&C_d1, sizeof(float)*SegSize);
    hipMalloc((void**)&A_d2, sizeof(float)*SegSize);
    hipMalloc((void**)&B_d2, sizeof(float)*SegSize);
    hipMalloc((void**)&C_d2, sizeof(float)*SegSize);



    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------


    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    unsigned int i = 0;

    for(i = 0;VecSize - i >= SegSize*StreamN; i += SegSize*StreamN){
        hipMemcpyAsync(A_d0, A_h+i, SegSize*sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(B_d0, B_h+i, SegSize*sizeof(float), hipMemcpyHostToDevice, stream0);

        hipMemcpyAsync(A_d1, A_h+i+SegSize, SegSize*sizeof(float), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(B_d1, B_h+i+SegSize, SegSize*sizeof(float), hipMemcpyHostToDevice, stream1);

        hipMemcpyAsync(A_d2, A_h+i+2*SegSize, SegSize*sizeof(float), hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(B_d2, B_h+i+2*SegSize, SegSize*sizeof(float), hipMemcpyHostToDevice, stream2);
        
        // Launch kernel  ---------------------------
        printf("Launching kernel..."); fflush(stdout);
        startTime(&timer);


        VecAdd<<<(SegSize - 1)/BLOCK_SIZE+1, BLOCK_SIZE, 0, stream0>>>(SegSize, A_d0, B_d0, C_d0);
        VecAdd<<<(SegSize - 1)/BLOCK_SIZE+1, BLOCK_SIZE, 0, stream1>>>(SegSize, A_d1, B_d1, C_d1);
        VecAdd<<<(SegSize - 1)/BLOCK_SIZE+1, BLOCK_SIZE, 0, stream2>>>(SegSize, A_d2, B_d2, C_d2);
        
        

        printf("Copying data from device to host..."); fflush(stdout);
        startTime(&timer);

        hipMemcpyAsync(C_h+i, C_d0, SegSize*sizeof(float), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(C_h+i+SegSize, C_d1, SegSize*sizeof(float), hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(C_h+i+2*SegSize, C_d2, SegSize*sizeof(float), hipMemcpyDeviceToHost, stream2);

        stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    }

    

    hipMalloc((void**)&A_d0, sizeof(float)*leftover);
    hipMalloc((void**)&B_d0, sizeof(float)*leftover);
    hipMalloc((void**)&C_d0, sizeof(float)*leftover);

    hipMemcpyAsync(A_d0, A_h + i, leftover*sizeof(float), hipMemcpyHostToDevice, stream0);
    hipMemcpyAsync(B_d0, B_h + i, leftover*sizeof(float), hipMemcpyHostToDevice, stream0);

    VecAdd<<<1, leftover, 0, stream0>>>(leftover, A_d0, B_d0, C_d0);

    hipMemcpyAsync(C_h+i, C_d0, leftover*sizeof(float), hipMemcpyDeviceToHost, stream0);


    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    

    verify(A_h, B_h, C_h, VecSize);


    // Free memory ------------------------------------------------------------

    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);

    hipFree(A_d0);
    hipFree(B_d0);
    hipFree(C_d0);

    hipFree(A_d1);
    hipFree(B_d1);
    hipFree(C_d1);

    hipFree(A_d2);
    hipFree(B_d2);
    hipFree(C_d2);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    //INSERT CODE HERE
    return 0;

}
