/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
    

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n){
        C[i] = A[i] + B[i];
    }
}


/*void basicVecAdd( float *A,  float *B, float *C, int n)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 512;

    unsigned int GRID_SIZE = ceil((double)n/(double)BLOCK_SIZE); 

    dim3 DimGrid(GRID_SIZE, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    VecAdd<<<DimGrid, DimBlock>>>(n, A, B, C);

    //INSERT CODE HERE

}*/

